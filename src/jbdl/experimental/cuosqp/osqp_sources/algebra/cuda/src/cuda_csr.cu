#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019-2021 ETH Zurich, Automatic Control Lab,
 *  Michel Schubiger, Goran Banjac.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cuda_csr.h"
#include "cuda_configure.h"
#include "cuda_handler.h"
#include "cuda_lin_alg.h"   /* --> cuda_vec_gather */
#include "cuda_malloc.h"
#include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */

#include "csr_type.h"
#include "glob_opts.h"

#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#ifdef __cplusplus
extern "C" {extern CUDA_Handle_t *CUDA_handle;}
#endif

/* This function is implemented in cuda_lin_alg.cu */
extern void scatter(c_float *out, const c_float *in, const c_int *ind, c_int n);


/*******************************************************************************
 *                            GPU Kernels                                      *
 *******************************************************************************/

 /*
 * Expand an upper triangular matrix given in COO format to a symmetric
 * matrix. Each entry is duplicated with its column- and row index switched.
 * In the case of a diagonal element we set the indices to a value  that is
 * larger than n to easily remove it later. This is done to keep the memory
 * patern one to one (MAP operation).
 * 
 * Additionally, it adds additional n diagonal elements to have a full 
 * diagonal.
 * 
 * The output arrays row_ind_out and col_ind_out have to be of size 2*nnz+n.
 */
__global__ void fill_full_matrix_kernel(c_int       *row_ind_out,
                                        c_int       *col_ind_out,
                                        c_int       *nnz_on_diag,
                                        c_int       *has_non_zero_diag_element,
                                        const c_int *__restrict__ row_ind_in,
                                        const c_int *__restrict__ col_ind_in,
                                        c_int        nnz,
                                        c_int        n) {

  c_int idx = threadIdx.x + blockDim.x * blockIdx.x;
  c_int grid_size = blockDim.x * gridDim.x;

  for(c_int i = idx; i < nnz; i += grid_size) {
    c_int row = row_ind_in[i];
    c_int column = col_ind_in[i];

    row_ind_out[i] = row;
    col_ind_out[i] = column;

    if (row == column) {
      has_non_zero_diag_element[row] = 1;
      row_ind_out[i + nnz] = column + n; /* dummy value for sorting and removal later on */
      col_ind_out[i + nnz] = row + n;
      atomicAdd(nnz_on_diag, 1);
    }
    else {
      row_ind_out[i + nnz] = column;
      col_ind_out[i + nnz] = row;
    }
  }
}

/**
 * Insert elements at structural zeros on the diagonal of the sparse matrix
 * specified by row and column index (COO format). To keep a one-to-one memory
 * patern we add n new elements to the matrix. In case where there already is a
 * diagonal element we add a dummy entry. The dummy entries will be removed later.
 */
__global__ void add_diagonal_kernel(c_int       *row_ind,
                                    c_int       *col_ind,
                                    const c_int *has_non_zero_diag_element,
                                    c_int        n) {

  c_int idx = threadIdx.x + blockDim.x * blockIdx.x;
  c_int grid_size = blockDim.x * gridDim.x;

  for(c_int row = idx; row < n; row += grid_size) {
    if (has_non_zero_diag_element[row] == 0) {
      row_ind[row] = row; 
      col_ind[row] = row;
    }
    else {
      row_ind[row] = row + n; /* dummy value, for easy removal after sorting */
      col_ind[row] = row + n;
    }
  }
}

/*
 * Permutation in: (size n, range 2*nnz+n):
 * 
 * Gathers from the following array to create the full matrix :
 * 
 *       |P_lower->val|P_lower->val|zeros(n)|
 *
 *       
 * Permutation out: (size n, range new_range)
 * 
 * Gathers from the following array to create the full matrix :
 * 
 *          |P_lower->val|zeros(1)|
 *                             
 *          | x[i] mod new_range    if x[i] <  2 * new_range
 * x[i] ->  | new_range             if x[i] >= 2 * new_range   
 * 
 */
__global__ void reduce_permutation_kernel(c_int *permutation,
                                          c_int  new_range,
                                          c_int  n) {

  c_int idx = threadIdx.x + blockDim.x * blockIdx.x;
  c_int grid_size = blockDim.x * gridDim.x;

  for(c_int i = idx; i < n; i += grid_size) {
    if (permutation[i] < 2 * new_range) {
      permutation[i] = permutation[i] % new_range;
    }
    else {
      permutation[i] = new_range; /* gets the 0 element at nnz+1 of the value array */
    }
  }
}

__global__ void get_diagonal_indices_kernel(c_int *row_ind,
                                            c_int *col_ind,
                                            c_int  nnz,
                                            c_int *diag_index) {

  c_int idx = threadIdx.x + blockDim.x * blockIdx.x;
  c_int grid_size = blockDim.x * gridDim.x;

  for (c_int index = idx; index < nnz; index += grid_size) {
    c_int row = row_ind[index];
    c_int column = col_ind[index];

    if (row == column) {
      diag_index[row] = index;
    }
  }
}

__global__ void predicate_generator_kernel(const c_int *row_ind,
                                           const c_int *row_predicate,
                                           c_int       *predicate,
                                           c_int        nnz) {

  c_int idx = threadIdx.x + blockDim.x * blockIdx.x;
  c_int grid_stride = gridDim.x * blockDim.x;

  for(c_int i = idx; i < nnz; i += grid_stride) {
    c_int row = row_ind[i];
    predicate[i] = row_predicate[row];
  }
}

template<typename T>
__global__ void compact(const T *data_in,
                        T       *data_out,
                        c_int   *predicate,
                        c_int   *scatter_addres,
                        c_int    n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if(idx < n) {
    if(predicate[idx]) {
      int write_ind = scatter_addres[idx] - 1;
      data_out[write_ind] = data_in[idx];
    }
  }
}

__global__ void compact_rows(const c_int *row_ind,
                             c_int       *data_out,
                             c_int       *new_row_number,
                             c_int       *predicate,
                             c_int       *scatter_addres,
                             c_int        n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if(idx < n) {
    if(predicate[idx]) {
      c_int write_ind = scatter_addres[idx] - 1;
      c_int row = row_ind[idx];
      data_out[write_ind] = new_row_number[row]-1;
    }
  }
}

__global__ void vector_init_abs_kernel(const c_int *a,
                                       c_int       *b,
                                       c_int        n) {

  c_int i  = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < n) {
    b[i] = abs(a[i]);
  }
}


/*******************************************************************************
 *                         Private Functions                                   *
 *******************************************************************************/

 /*
 *  Update the size of buffer used for the merge path based
 *  sparse matrix-vector product (spmv).
 */
void update_mp_buffer(csr *P) {

  size_t bufferSizeInBytes = 0;
  c_float alpha = 1.0;

  checkCudaErrors(cusparseCsrmvEx_bufferSize(CUDA_handle->cusparseHandle, P->alg,
                                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                             P->m, P->n, P->nnz, &alpha,
                                             CUDA_FLOAT, P->MatDescription, P->val,
                                             CUDA_FLOAT, P->row_ptr, P->col_ind, NULL,
                                             CUDA_FLOAT, &alpha, CUDA_FLOAT, NULL,
                                             CUDA_FLOAT, CUDA_FLOAT, &bufferSizeInBytes));
  
  if (bufferSizeInBytes > P->bufferSizeInBytes) {
    cuda_free((void **) &P->buffer);                            
    cuda_malloc((void **) &P->buffer, bufferSizeInBytes);
    P->bufferSizeInBytes = bufferSizeInBytes;
  }
}

 /*
 *  Creates a CSR matrix with the specified dimension (m,n,nnz).
 *  
 *  If specified, it allocates proper amount of device memory
 *  allocate_on_device = 1: device memory for CSR
 *  allocate_on_device = 2: device memory for CSR (+ col_ind)  
 */
csr* csr_alloc(c_int m,
               c_int n,
               c_int nnz,
               c_int allocate_on_device) {

  csr *dev_mat = (csr*) c_calloc(1, sizeof(csr));

  if (!dev_mat) return NULL;

  dev_mat->m   = m;
  dev_mat->n   = n;
  dev_mat->nnz = nnz;
  
#if defined IS_WINDOWS && __CUDACC_VER_MAJOR__ < 11
  // MERGE_PATH is not working properly on WINDOWS
  dev_mat->alg = CUSPARSE_ALG_NAIVE;
#else
  // Since CUDA 11 there is only one algorithm
  dev_mat->alg = CUSPARSE_ALG_MERGE_PATH;
#endif

  dev_mat->buffer = NULL;
  dev_mat->bufferSizeInBytes = 0;

  checkCudaErrors(hipsparseCreateMatDescr(&dev_mat->MatDescription));
  hipsparseSetMatType(dev_mat->MatDescription, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(dev_mat->MatDescription, HIPSPARSE_INDEX_BASE_ZERO);

  if (allocate_on_device > 0) {
    cuda_calloc((void **) &dev_mat->val, (dev_mat->nnz + 1) * sizeof(c_float));
    cuda_malloc((void **) &dev_mat->row_ptr, (dev_mat->m + 1) * sizeof(c_int)); 
    cuda_malloc((void **) &dev_mat->col_ind, dev_mat->nnz * sizeof(c_int));

    if (allocate_on_device > 1) {
      cuda_malloc((void **) &dev_mat->row_ind, dev_mat->nnz * sizeof(c_int));
    } 
  }
  return dev_mat;
}

/*
 *  Copy CSR matrix from host to device.
 *  The device memory should be pre-allocated.
 */
void csr_copy_h2d(csr           *dev_mat,
                  const c_int   *h_row_ptr,
                  const c_int   *h_col_ind,
                  const c_float *h_val) {

  checkCudaErrors(hipMemcpy(dev_mat->row_ptr, h_row_ptr, (dev_mat->m + 1) * sizeof(c_int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev_mat->col_ind, h_col_ind, dev_mat->nnz * sizeof(c_int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev_mat->val, h_val, dev_mat->nnz * sizeof(c_float), hipMemcpyHostToDevice));
}

csr* csr_init(c_int          m,
              c_int          n,
              const c_int   *h_row_ptr,
              const c_int   *h_col_ind,
              const c_float *h_val) {
    
  csr *dev_mat = csr_alloc(m, n, h_row_ptr[m], 1);
  
  if (!dev_mat) return NULL;
  
  if (m == 0) return dev_mat;

  /* copy_matrix_to_device */
  csr_copy_h2d(dev_mat, h_row_ptr, h_col_ind, h_val);
  update_mp_buffer(dev_mat);

  return dev_mat;
}

/*
 *  Compress row indices from the COO format to the row pointer
 *  of the CSR format.
 */
void compress_row_ind(csr *mat) {

  cuda_free((void** ) &mat->row_ptr);
  cuda_malloc((void** ) &mat->row_ptr, (mat->m + 1) * sizeof(c_float));
  checkCudaErrors(hipsparseXcoo2csr(CUDA_handle->cusparseHandle, mat->row_ind, mat->nnz, mat->m, mat->row_ptr, HIPSPARSE_INDEX_BASE_ZERO));
}

void csr_expand_row_ind(csr *mat) {

  if (!mat->row_ind) {
    cuda_malloc((void** ) &mat->row_ind, mat->nnz * sizeof(c_float));
    checkCudaErrors(hipsparseXcsr2coo(CUDA_handle->cusparseHandle, mat->row_ptr, mat->nnz, mat->m, mat->row_ind, HIPSPARSE_INDEX_BASE_ZERO));
  }
}

/*
 *  Sorts matrix in COO format by row. It returns a permutation
 *  vector that describes reordering of the elements.
 */
c_int* coo_sort(csr *A) {

  c_int *A_to_At_permutation;
  char *pBuffer;
  size_t pBufferSizeInBytes;

  cuda_malloc((void **) &A_to_At_permutation, A->nnz * sizeof(c_int));
  checkCudaErrors(hipsparseCreateIdentityPermutation(CUDA_handle->cusparseHandle, A->nnz, A_to_At_permutation));

  checkCudaErrors(hipsparseXcoosort_bufferSizeExt(CUDA_handle->cusparseHandle, A->m, A->n, A->nnz, A->row_ind, A->col_ind, &pBufferSizeInBytes));

  cuda_malloc((void **) &pBuffer, pBufferSizeInBytes * sizeof(char));

  checkCudaErrors(hipsparseXcoosortByRow(CUDA_handle->cusparseHandle, A->m, A->n, A->nnz, A->row_ind, A->col_ind, A_to_At_permutation, pBuffer));

  cuda_free((void **) &pBuffer);

  return A_to_At_permutation;
}

/*
 * Compute transpose of a matrix in COO format.
 */
void coo_tranpose(csr* A) {
  c_int m = A->m;
  A->m = A->n;
  A->n = m;

  c_int *row_ind = A->row_ind;
  A->row_ind = A->col_ind;
  A->col_ind = row_ind;
}

/*
 *  values[i] = values[permutation[i]] for i in [0,n-1]
 */
void permute_vector(c_float     *values,
                    const c_int *permutation,
                    c_int        n) {

  c_float *permuted_values;
  cuda_malloc((void **) &permuted_values, n * sizeof(c_float));

  cuda_vec_gather(n, values, permuted_values, permutation);

  checkCudaErrors(hipMemcpy(values, permuted_values, n * sizeof(c_float), hipMemcpyDeviceToDevice));
  cuda_free((void **) &permuted_values);
}

/*
 *  target[i] = source[permutation[i]] for i in [0,n-1]
 *  
 *  target and source cannot point to the same location
 */
void permute_vector(c_float       *target,
                    const c_float *source,
                    const c_int   *permutation,
                    c_int          n) {

  cuda_vec_gather(n, source, target, permutation);
}

/*
 *  Copy the values and pointers form target to the source matrix.
 *  The device memory of source has to be freed first to avoid a
 *  memory leak in case it holds allocated memory.
 *  
 *  The MatrixDescription has to be destroyed first since it is a
 *  pointer hidded by a typedef.
 *  
 *  The pointers of source matrix are set to NULL to avoid
 *  accidental freeing of the associated memory blocks.
 */
void copy_csr(csr* target,
              csr* source) {

  target->m                 = source->m;
  target->n                 = source->n;
  target->nnz               = source->nnz;
  target->bufferSizeInBytes = source->bufferSizeInBytes;
  target->alg               = source->alg;

  hipsparseDestroyMatDescr(target->MatDescription);
  cuda_free((void **) &target->val);
  cuda_free((void **) &target->row_ind);
  cuda_free((void **) &target->row_ptr);
  cuda_free((void **) &target->col_ind);
  cuda_free((void **) &target->buffer);

  target->val            = source->val;
  target->row_ind        = source->row_ind;
  target->row_ptr        = source->row_ptr;
  target->col_ind        = source->col_ind;
  target->buffer         = source->buffer;
  target->MatDescription = source->MatDescription; 

  source->val            = NULL;
  source->row_ind        = NULL;
  source->row_ptr        = NULL;
  source->col_ind        = NULL;
  source->buffer         = NULL;
  source->MatDescription = NULL;
}

void csr_triu_to_full(csr    *P_triu,
                      c_int **P_triu_to_full_permutation,
                      c_int **P_diag_indices) {

  c_int number_of_blocks;
  c_int *has_non_zero_diag_element, *d_nnz_diag;
  c_int h_nnz_diag, Full_nnz, nnz_triu, n, nnz_max_Full;
  c_int offset;

  nnz_triu     = P_triu->nnz;
  n            = P_triu->n;
  nnz_max_Full = 2*nnz_triu + n;

  csr *Full_P = csr_alloc(n, n, nnz_max_Full, 2);
  cuda_calloc((void **) &has_non_zero_diag_element, n * sizeof(c_int));
  cuda_calloc((void **) &d_nnz_diag, sizeof(c_int));

  csr_expand_row_ind(P_triu);

  number_of_blocks = (nnz_triu / THREADS_PER_BLOCK) + 1;
  fill_full_matrix_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(Full_P->row_ind, Full_P->col_ind, d_nnz_diag, has_non_zero_diag_element, P_triu->row_ind, P_triu->col_ind, nnz_triu, n);

  offset = 2 * nnz_triu;
  number_of_blocks = (n / THREADS_PER_BLOCK) + 1;
  add_diagonal_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(Full_P->row_ind + offset, Full_P->col_ind + offset, has_non_zero_diag_element, n);

  /* The Full matrix now is of size (2n)x(2n)
    *                  [P 0]
    *                  [0 D]
    * where P is the desired full matrix and D is
    * a diagonal that contains dummy values
  */
  
  checkCudaErrors(hipMemcpy(&h_nnz_diag, d_nnz_diag, sizeof(c_int), hipMemcpyDeviceToHost));

  Full_nnz = (2 * (nnz_triu - h_nnz_diag)) + n;
  c_int *d_P = coo_sort(Full_P);

  number_of_blocks = (nnz_triu / THREADS_PER_BLOCK) + 1;
  reduce_permutation_kernel<<<number_of_blocks,THREADS_PER_BLOCK>>>(d_P, nnz_triu, Full_nnz);

  permute_vector(Full_P->val, P_triu->val, d_P, Full_nnz);

  cuda_malloc((void **) P_triu_to_full_permutation, Full_nnz * sizeof(c_int));
  checkCudaErrors(hipMemcpy(*P_triu_to_full_permutation, d_P, Full_nnz * sizeof(c_int), hipMemcpyDeviceToDevice));
  cuda_malloc((void **) P_diag_indices, n * sizeof(c_int));

  number_of_blocks = (Full_nnz / THREADS_PER_BLOCK) + 1;
  get_diagonal_indices_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(Full_P->row_ind, Full_P->col_ind, Full_nnz, *P_diag_indices);

  Full_P->nnz = Full_nnz;
  compress_row_ind(Full_P);
  update_mp_buffer(Full_P); 
  copy_csr(P_triu, Full_P);

  cuda_mat_free(Full_P);
  cuda_free((void **) &d_P);
  cuda_free((void **) &d_nnz_diag);
  cuda_free((void **) &has_non_zero_diag_element);
}

/**
 * Matrix A is converted from CSC to CSR. The data in A is interpreted as
 * being in CSC format, even if it is in CSR.
 * This operation is equivalent to a transpose. We temporarily allocate space
 * for the new matrix since this operation cannot be done inplace.
 * Additionally, a gather indices vector is generated to perform the conversion
 * from A to A' faster during a matrix update.
 */
void csr_transpose(csr    *A,
                   c_int **A_to_At_permutation) {

  (*A_to_At_permutation) = NULL;

  if (A->nnz == 0) {
    c_int tmp = A->n;
    A->n = A->m;
    A->m = tmp;
    return;
  }

  csr_expand_row_ind(A);
  coo_tranpose(A);
  (*A_to_At_permutation) = coo_sort(A);
  compress_row_ind(A);

  permute_vector(A->val, *A_to_At_permutation, A->nnz);

  update_mp_buffer(A);
}


/*******************************************************************************
 *                           API Functions                                     *
 *******************************************************************************/

void cuda_mat_init_P(const csc  *mat,
                     csr       **P,
                     c_float   **d_P_triu_val,
                     c_int     **d_P_triu_to_full_ind,
                     c_int     **d_P_diag_ind) {

  c_int n   = mat->n;
  c_int nnz = mat->p[n];
  
  /* Initialize upper triangular part of P */
  *P = csr_init(n, n, mat->p, mat->i, mat->x);

  /* Convert P to a full matrix. Store indices of diagonal and triu elements. */
  csr_triu_to_full(*P, d_P_triu_to_full_ind, d_P_diag_ind);
  csr_expand_row_ind(*P);

  /* We need 0.0 at val[nzz] -> nnz+1 elements */
  cuda_calloc((void **) d_P_triu_val, (nnz+1) * sizeof(c_float));

  /* Store triu elements */
  checkCudaErrors(hipMemcpy(*d_P_triu_val, mat->x, nnz * sizeof(c_float), hipMemcpyHostToDevice));
}

void cuda_mat_init_A(const csc  *mat,
                     csr       **A,
                     csr       **At,
                     c_int     **d_A_to_At_ind) {

  c_int m = mat->m;
  c_int n = mat->n;

  /* Initializing At is easy since it is equal to A in CSC */
  *At = csr_init(n, m, mat->p, mat->i, mat->x);
  csr_expand_row_ind(*At);

  /* We need to take transpose of At to get A */
  *A = csr_init(n, m, mat->p, mat->i, mat->x);
  csr_transpose(*A, d_A_to_At_ind);
  csr_expand_row_ind(*A);
}

void cuda_mat_update_P(const c_float  *Px,
                       const c_int    *Px_idx,
                       c_int           Px_n,
                       csr           **P,
                       c_float        *d_P_triu_val,
                       c_int          *d_P_triu_to_full_ind,
                       c_int          *d_P_diag_ind,
                       c_int           P_triu_nnz) {

  if (!Px_idx) { /* Update whole P */
    c_float *d_P_val_new;

    /* Allocate memory */
    cuda_malloc((void **) &d_P_val_new, (P_triu_nnz + 1) * sizeof(c_float));

    /* Copy new values from host to device */
    checkCudaErrors(hipMemcpy(d_P_val_new, Px, P_triu_nnz * sizeof(c_float), hipMemcpyHostToDevice));

    cuda_vec_gather((*P)->nnz, d_P_val_new, (*P)->val, d_P_triu_to_full_ind);

    cuda_free((void **) &d_P_val_new);
  }
  else { /* Update P partially */
    c_float *d_P_val_new;
    c_int   *d_P_ind_new;

    /* Allocate memory */
    cuda_malloc((void **) &d_P_val_new, Px_n * sizeof(c_float));
    cuda_malloc((void **) &d_P_ind_new, Px_n * sizeof(c_int));

    /* Copy new values and indices from host to device */
    checkCudaErrors(hipMemcpy(d_P_val_new, Px,     Px_n * sizeof(c_float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_P_ind_new, Px_idx, Px_n * sizeof(c_int),   hipMemcpyHostToDevice));

    /* Update d_P_triu_val */
    scatter(d_P_triu_val, d_P_val_new, d_P_ind_new, Px_n);

    /* Gather from d_P_triu_val to update full P */
    cuda_vec_gather((*P)->nnz, d_P_triu_val, (*P)->val, d_P_triu_to_full_ind);

    cuda_free((void **) &d_P_val_new);
    cuda_free((void **) &d_P_ind_new);
  }
}

void cuda_mat_update_A(const c_float  *Ax,
                       const c_int    *Ax_idx,
                       c_int           Ax_n,
                       csr           **A,
                       csr           **At,
                       c_int          *d_A_to_At_ind) {

  c_int Annz     = (*A)->nnz;
  c_float *Aval  = (*A)->val;
  c_float *Atval = (*At)->val;

  if (!Ax_idx) { /* Update whole A */
    /* Updating At is easy since it is equal to A in CSC */
    checkCudaErrors(hipMemcpy(Atval, Ax, Annz * sizeof(c_float), hipMemcpyHostToDevice));

    /* Updating A requires transpose of A_new */
    cuda_vec_gather(Annz, Atval, Aval, d_A_to_At_ind);
  }
  else { /* Update A partially */
    c_float *d_At_val_new;
    c_int   *d_At_ind_new;

    /* Allocate memory */
    cuda_malloc((void **) &d_At_val_new, Ax_n * sizeof(c_float));
    cuda_malloc((void **) &d_At_ind_new, Ax_n * sizeof(c_int));

    /* Copy new values and indices from host to device */
    checkCudaErrors(hipMemcpy(d_At_val_new, Ax,     Ax_n * sizeof(c_float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_At_ind_new, Ax_idx, Ax_n * sizeof(c_int),   hipMemcpyHostToDevice));

    /* Update At first since it is equal to A in CSC */
    scatter(Atval, d_At_val_new, d_At_ind_new, Ax_n);

    cuda_free((void **) &d_At_val_new);
    cuda_free((void **) &d_At_ind_new);

    /* Gather from Atval to construct Aval */
    cuda_vec_gather(Annz, Atval, Aval, d_A_to_At_ind);
  }
}

void cuda_mat_free(csr *mat) {
  if (mat) {
    cuda_free((void **) &mat->val);
    cuda_free((void **) &mat->row_ptr);
    cuda_free((void **) &mat->col_ind);
    cuda_free((void **) &mat->buffer);
    cuda_free((void **) &mat->row_ind);
    hipsparseDestroyMatDescr(mat->MatDescription);
    c_free(mat);
  }
}

void cuda_submat_byrows(const csr    *A,
                        const c_int  *d_rows,
                        csr         **Ared,
                        csr         **Aredt) {

  c_int new_m = 0;

  c_int n   = A->n;
  c_int m   = A->m;
  c_int nnz = A->nnz;

  c_int *d_predicate;
  c_int *d_compact_address;
  c_int *d_row_predicate;
  c_int *d_new_row_number;

  cuda_malloc((void **) &d_row_predicate,  m * sizeof(c_int));
  cuda_malloc((void **) &d_new_row_number, m * sizeof(c_int));

  cuda_malloc((void **) &d_predicate,       nnz * sizeof(c_int));
  cuda_malloc((void **) &d_compact_address, nnz * sizeof(c_int));

  // Copy rows array to device and set -1s to ones
  checkCudaErrors(hipMemcpy(d_row_predicate, d_rows, m * sizeof(c_int), hipMemcpyDeviceToDevice));
  vector_init_abs_kernel<<<(m/THREADS_PER_BLOCK) + 1,THREADS_PER_BLOCK>>>(d_row_predicate, d_row_predicate, m);

  // Calculate new row numbering and get new number of rows
  thrust::inclusive_scan(thrust::device, d_row_predicate, d_row_predicate + m, d_new_row_number);
  if (m) {
    checkCudaErrors(hipMemcpy(&new_m, &d_new_row_number[m-1], sizeof(c_int), hipMemcpyDeviceToHost));
  }
  else {
    (*Ared) = (csr *) c_calloc(1, sizeof(csr));
    (*Ared)->n = n;

    (*Aredt) = (csr *) c_calloc(1, sizeof(csr));
    (*Aredt)->m = n;

    return;
  }

  // Generate predicates per element from per row predicate
  predicate_generator_kernel<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->row_ind, d_row_predicate, d_predicate, nnz);

  // Get array offset for compacting and new nnz
  thrust::inclusive_scan(thrust::device, d_predicate, d_predicate + nnz, d_compact_address);
  c_int nnz_new;
  if (nnz) checkCudaErrors(hipMemcpy(&nnz_new, &d_compact_address[nnz-1], sizeof(c_int), hipMemcpyDeviceToHost));

  // allocate new matrix (2 -> allocate row indices as well)
  (*Ared) = csr_alloc(new_m, n, nnz_new, 2);

  // Compact arrays according to given predicates, special care has to be taken for the rows
  compact_rows<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->row_ind, (*Ared)->row_ind, d_new_row_number, d_predicate, d_compact_address, nnz);
  compact<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->col_ind, (*Ared)->col_ind, d_predicate, d_compact_address, nnz);
  compact<<<(nnz/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->val, (*Ared)->val, d_predicate, d_compact_address, nnz);

  // Generate row pointer
  compress_row_ind(*Ared);

  // Update merge path buffer (CsrmvEx)
  update_mp_buffer(*Ared);

  // We first make a copy of Ared
  *Aredt = csr_alloc(new_m, n, nnz_new, 1);
  checkCudaErrors(hipMemcpy((*Aredt)->val,     (*Ared)->val,     nnz_new   * sizeof(c_float), hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy((*Aredt)->row_ptr, (*Ared)->row_ptr, (new_m+1) * sizeof(c_int),   hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy((*Aredt)->col_ind, (*Ared)->col_ind, nnz_new   * sizeof(c_int),   hipMemcpyDeviceToDevice));

  c_int *d_A_to_At_ind;
  csr_transpose(*Aredt, &d_A_to_At_ind);

  // Update merge path buffer (CsrmvEx)
  update_mp_buffer(*Aredt);

  cuda_free((void**)&d_A_to_At_ind);
  cuda_free((void**)&d_predicate);
  cuda_free((void**)&d_compact_address);
  cuda_free((void**)&d_row_predicate);
  cuda_free((void**)&d_new_row_number);
}

